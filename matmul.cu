#define TILE_SIZE 4
#define KBLOCK 8
#define TILE_SIZEB 8
#define BLOCK_SIZEB 128
#define BLOCK_SIZE 128
#define KBLOCKB 8
#define STRIDE BLOCK_SIZEB/TILE_SIZEB
#define M 2560
#define N 2048
#define K 2048
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
/**
Algorithm 0: The most naive CUDA matmul kernel.
The result serves as the baseline.
*/
__global__ void matmul_0(float *A, float *B, float *C)
{
	int m_i = blockIdx.y * blockDim.y + threadIdx.y;
	int n_i = blockIdx.x * blockDim.x + threadIdx.x;
	float c = 0;
	for(int i=0; i<K; i++){
		c += A[m_i*K + i]*B[i*N + n_i];
	}
	if(m_i < M && n_i < N){
		C[m_i*N + n_i] = c;
    }
}

__global__ void matmul_1(float *A, float *B, float *C)
{
	int m_i = blockIdx.y * BLOCK_SIZE + threadIdx.y * TILE_SIZE;
	int n_i = blockIdx.x * BLOCK_SIZE + threadIdx.x * TILE_SIZE;
    float a[TILE_SIZE][KBLOCK];
    float b[KBLOCK][TILE_SIZE];
	float c[TILE_SIZE][TILE_SIZE];
    for(int i=0; i<TILE_SIZE; i++){
        for(int j=0; j<TILE_SIZE; j++){
            c[i][j]=0;
        }
    }
	for(int k_i=0; k_i<K; k_i += TILE_SIZE){
		for(int i=0; i<TILE_SIZE; i++){
            for(int k=0; k<KBLOCK; k++){
                a[i][k] = A[(m_i+i)*K + k_i + k];
                b[k][i] = B[(k_i + k)*N + n_i + i];
            }
        }
        
        for(int i=0; i<TILE_SIZE; i++){
            for(int j=0; j<TILE_SIZE; j++){
                #pragma unroll
                for(int k=0; k<KBLOCK; k++)
                    c[i][j] += a[i][k] * b[k][j];
            }
        }
	}
    for(int i=0; i<TILE_SIZE; i++){
        for(int j=0; j<TILE_SIZE; j++){
            if(m_i+i < M && n_i+j < N){
                C[(m_i+i)*N + n_i + j] = c[i][j];
            }
        }
    }

}

__global__ void matmul_2(float *A, float *B, float *C)
{
    int m_b = blockIdx.y * BLOCK_SIZE;
    int n_b = blockIdx.x * BLOCK_SIZE;
    int m_t = threadIdx.y * TILE_SIZE;
    int n_t = threadIdx.x * TILE_SIZE;
	int m_i = m_b + m_t;
	int n_i = n_b + n_t;
    float a[TILE_SIZE][KBLOCK];
    float b[KBLOCK][TILE_SIZE];
	float c[TILE_SIZE][TILE_SIZE];
    __shared__ float sa[BLOCK_SIZE*KBLOCK];
    __shared__ float sb[KBLOCK*BLOCK_SIZE];
    
    for(int i=0; i<TILE_SIZE; i++){
        for(int j=0; j<TILE_SIZE; j++){
            c[i][j]=0;
        }
    }
    
    
	for(int k_i=0; k_i<K; k_i += KBLOCK){
        
        for (int t=threadIdx.y*blockDim.x + threadIdx.x; t<BLOCK_SIZE*KBLOCK; t += blockDim.x*blockDim.y){
            sa[t] = A[(m_b + t/KBLOCK)*K + k_i + t%KBLOCK];
            sb[t] = B[(k_i + t/BLOCK_SIZE)*N + n_b + t%BLOCK_SIZE];
        }
        __syncthreads();
		for(int i=0; i<TILE_SIZE; i++){
            for(int k=0; k<KBLOCK; k++){
                a[i][k] = sa[(m_t + i)*KBLOCK + k];
                b[k][i] = sb[k*BLOCK_SIZE + n_t + i];
            }
        }
        
        for(int i=0; i<TILE_SIZE; i++){
            for(int j=0; j<TILE_SIZE; j++){
                #pragma unroll
                for(int k=0; k<KBLOCK; k++)
                    c[i][j] += a[i][k] * b[k][j];
            }
        }
        __syncthreads();
	}
    for(int i=0; i<TILE_SIZE; i++){
        for(int j=0; j<TILE_SIZE; j++){
            if(m_i+i < M && n_i+j < N){
                C[(m_i+i)*N + n_i + j] = c[i][j];
            }
        }
    }

}

__global__ void matmul_3(float *A, float *B, float *C)
{
    int m_b = blockIdx.y * BLOCK_SIZEB;
    int n_b = blockIdx.x * BLOCK_SIZEB;
    int m_t = threadIdx.y * TILE_SIZEB;
    int n_t = threadIdx.x * TILE_SIZEB;
	int m_i = m_b + m_t;
	int n_i = n_b + n_t;
    float a[TILE_SIZEB];
    float b[TILE_SIZEB];
	float c[TILE_SIZEB][TILE_SIZEB];
    __shared__ float sa[BLOCK_SIZEB*KBLOCKB];
    __shared__ float sb[KBLOCKB*BLOCK_SIZEB];
    
    for(int i=0; i<TILE_SIZEB; i++){
        for(int j=0; j<TILE_SIZEB; j++){
            c[i][j]=0;
        }
    }
    
    
	for(int k_i=0; k_i<K; k_i += KBLOCKB){
        
        for (int t=threadIdx.y*blockDim.x + threadIdx.x; t<BLOCK_SIZEB*KBLOCKB; t += blockDim.x*blockDim.y){
            sa[t] = A[(m_b + t/KBLOCKB)*K + k_i + t%KBLOCKB];
            sb[t] = B[(k_i + t/BLOCK_SIZEB)*N + n_b + t%BLOCK_SIZEB];
        }
        __syncthreads();
        for(int k=0; k<KBLOCKB; k++){
            #pragma unroll
            for(int i=0; i<TILE_SIZEB; i++){
                a[i] = sa[(m_t + i)*KBLOCKB + k];
                b[i] = sb[k*BLOCK_SIZEB + n_t + i];
            }
            
            for(int i=0; i<TILE_SIZEB; i++){
                #pragma unroll
                for(int j=0; j<TILE_SIZEB; j++){
                    c[i][j] += a[i] * b[j];
                }
            }
        }
        __syncthreads();
	}
    for(int i=0; i<TILE_SIZEB; i++){
        for(int j=0; j<TILE_SIZEB; j++){
            if(m_i+i < M && n_i+j < N){
                C[(m_i+i)*N + n_i + j] = c[i][j];
            }
        }
    }

}

__global__ void matmul_4(float *A, float *B, float *C)
{
    int m_b = blockIdx.y * BLOCK_SIZEB;
    int n_b = blockIdx.x * BLOCK_SIZEB;

	int m_i = m_b + threadIdx.y;
	int n_i = n_b + threadIdx.x;
    float a[TILE_SIZEB];
    float b[TILE_SIZEB];
	float c[TILE_SIZEB][TILE_SIZEB];
    __shared__ float sa[BLOCK_SIZEB*KBLOCKB];
    __shared__ float sb[KBLOCKB*BLOCK_SIZEB];
    
    
    for(int i=0; i<TILE_SIZEB; i++){
        for(int j=0; j<TILE_SIZEB; j++){
            c[i][j]=0;
        }
    }
    
    
	for(int k_i=0; k_i<K; k_i += KBLOCKB){
        
        for (int t=threadIdx.y*STRIDE + threadIdx.x; t<BLOCK_SIZEB*KBLOCKB; t += STRIDE*STRIDE){
            sa[t] = A[(m_b + t/KBLOCKB)*K + k_i + t%KBLOCKB];
            sb[t] = B[(k_i + t/BLOCK_SIZEB)*N + n_b + t%BLOCK_SIZEB];
        }
        __syncthreads();
        for(int k=0; k<KBLOCKB; k++){
            #pragma unroll
            for(int i=0; i<TILE_SIZEB; i++){
                a[i] = sa[(i*STRIDE + threadIdx.y)*KBLOCKB + k];
                b[i] = sb[k*BLOCK_SIZEB + i*STRIDE + threadIdx.x];
            }
            
            for(int i=0; i<TILE_SIZEB; i++){
                #pragma unroll
                for(int j=0; j<TILE_SIZEB; j++){
                    c[i][j] += a[i] * b[j];
                }
            }
        }
        __syncthreads();
	}
    for(int i=0; i<TILE_SIZEB; i++){
        for(int j=0; j<TILE_SIZEB; j++){
            if(m_i+i*STRIDE < M && n_i+j*STRIDE < N){
                C[(m_i+i*STRIDE)*N + n_i + j*STRIDE] = c[i][j];
            }
        }
    }

}

__global__ void matmul_5(float *A, float *B, float *C)
{
    int m_b = blockIdx.y * BLOCK_SIZEB;
    int n_b = blockIdx.x * BLOCK_SIZEB;

	int m_i = m_b + threadIdx.y;
	int n_i = n_b + threadIdx.x;
    float a[TILE_SIZEB][KBLOCKB];
    float b[KBLOCKB][TILE_SIZEB];
	float c[TILE_SIZEB][TILE_SIZEB];
    __shared__ float sa[BLOCK_SIZEB*KBLOCKB];
    __shared__ float sb[KBLOCKB*BLOCK_SIZEB];
    
    for(int i=0; i<TILE_SIZEB; i++){
        for(int j=0; j<TILE_SIZEB; j++){
            c[i][j]=0;
        }
    }
    
    
	for(int k_i=0; k_i<K; k_i += KBLOCKB){
        
        for (int t=threadIdx.y*STRIDE + threadIdx.x; t<BLOCK_SIZEB*KBLOCKB; t += STRIDE*STRIDE){
            sa[t] = A[(m_b + t/KBLOCKB)*K + k_i + t%KBLOCKB];
            sb[t] = B[(k_i + t/BLOCK_SIZEB)*N + n_b + t%BLOCK_SIZEB];
        }
        __syncthreads();
		for(int i=0; i<TILE_SIZEB; i++){
            for(int k=0; k<KBLOCKB; k++){
                a[i][k] = sa[(i*STRIDE + threadIdx.y)*KBLOCKB + k];
                b[k][i] = sb[k*BLOCK_SIZEB + i*STRIDE + threadIdx.x];
            }
        }
        
        for(int i=0; i<TILE_SIZEB; i++){
            for(int j=0; j<TILE_SIZEB; j++){
                #pragma unroll
                for(int k=0; k<KBLOCKB; k++)
                    c[i][j] += a[i][k] * b[k][j];
            }
        }
        __syncthreads();
	}
    for(int i=0; i<TILE_SIZEB; i++){
        for(int j=0; j<TILE_SIZEB; j++){
            if(m_i+i*STRIDE < M && n_i+j*STRIDE < N){
                C[(m_i+i*STRIDE)*N + n_i + j*STRIDE] = c[i][j];
            }
        }
    }

}

int main(int argc, char *argv[])
{
	srand(0);
	float *cpu_a = (float*)malloc(sizeof(float)*M*K);
	float *cpu_b = (float*)malloc(sizeof(float)*K*N);
	float *cpu_c = (float*)malloc(sizeof(float)*M*N);
	float *cpu_c2 = (float*)malloc(sizeof(float)*M*N);

	for (int i=0; i<M*K; i++)
		cpu_a[i] = rand()/65536;

	for (int i=0; i<N*K; i++)
		cpu_b[i] = rand()/65536;

	//cudaEvent_t start, end;
	//cudaEventCreate(&start);
	//cudeEventCreate(&end);

	float *gpu_a, *gpu_b, *gpu_c;
	hipMalloc((void**)&gpu_a, sizeof(float)*M*K);
	hipMalloc((void**)&gpu_b, sizeof(float)*N*K);
	hipMalloc((void**)&gpu_c, sizeof(float)*M*N);
	
	hipMemcpy(gpu_a, cpu_a, sizeof(float)*M*K, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, cpu_b, sizeof(float)*N*K, hipMemcpyHostToDevice);

	dim3 grid0(M/TILE_SIZE,N/TILE_SIZE);
    dim3 block0(TILE_SIZE, TILE_SIZE);

	matmul_0<<<grid0, block0>>>(gpu_a, gpu_b, gpu_c);
    hipDeviceSynchronize();
    printf("%s\n",hipGetErrorString(hipPeekAtLastError()));
	hipMemcpy(cpu_c2, gpu_c, sizeof(float)*M*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
	//cudaEventRecord(start, 0);
    dim3 grid(M/BLOCK_SIZE,N/BLOCK_SIZE);
    dim3 block(BLOCK_SIZE/TILE_SIZE, BLOCK_SIZE/TILE_SIZE);

	matmul_2<<<grid, block>>>(gpu_a, gpu_b, gpu_c);
    hipDeviceSynchronize();
    printf("%s\n",hipGetErrorString(hipPeekAtLastError()));
	hipMemcpy(cpu_c, gpu_c, sizeof(float)*M*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    matmul_1<<<grid, block>>>(gpu_a, gpu_b, gpu_c);
    hipDeviceSynchronize();
    printf("%s\n",hipGetErrorString(hipPeekAtLastError()));
	hipMemcpy(cpu_c, gpu_c, sizeof(float)*M*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
	
    dim3 gridb(M/BLOCK_SIZEB,N/BLOCK_SIZEB);
    dim3 blockb(BLOCK_SIZEB/TILE_SIZEB, BLOCK_SIZEB/TILE_SIZEB);

    matmul_4<<<gridb, blockb>>>(gpu_a, gpu_b, gpu_c);
    hipDeviceSynchronize();
    printf("%s\n",hipGetErrorString(hipPeekAtLastError()));
	hipMemcpy(cpu_c, gpu_c, sizeof(float)*M*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
	matmul_5<<<gridb, blockb>>>(gpu_a, gpu_b, gpu_c);
    hipDeviceSynchronize();
    printf("%s\n",hipGetErrorString(hipPeekAtLastError()));
	hipMemcpy(cpu_c, gpu_c, sizeof(float)*M*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    float sum = 0.0;
    for (int i=0; i<M*N; i++)
		sum += cpu_c2[i];
    printf("%f\n",sum);
    sum = 0.0;
    for (int i=0; i<M*N; i++)
		sum += cpu_c[i];
    printf("%f\n",sum);
    sum = 0.0;
    for (int i=0; i<M*N; i++)
		sum += cpu_c[i] - cpu_c2[i];
    printf("%f\n",sum);
	//cudaEventRecord(end,0);
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
	free(cpu_a);
	free(cpu_b);
	free(cpu_c);
    free(cpu_c2);
}
